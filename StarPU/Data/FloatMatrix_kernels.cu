#include <hip/hip_runtime.h>

/********** ADD **********/

__global__ void FloatMatrixAdd(unsigned w, unsigned h, const float* A, unsigned ldA, const float* B, unsigned ldB, float* C, unsigned ldC) {
  unsigned gx = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned gy = blockDim.y * blockIdx.y + threadIdx.y;
  if (gx < w && gy < h)
    C[gy*ldC + gx] = A[gy*ldA + gx] + B[gy*ldB + gx];
}

extern "C" void cuda_floatmatrix_add(unsigned w, unsigned h, const float* A, unsigned ldA, const float* B, unsigned ldB, float* C, unsigned ldC) {
  dim3 grid((w + 15) / 15, (h + 15) / 15, 1);
  dim3 block(16,16,1);
  FloatMatrixAdd<<<grid,block>>>(w, h, A, ldA, B, ldB, C, ldC);
}

/********** SUB **********/

__global__ void FloatMatrixSub(unsigned w, unsigned h, const float* A, unsigned ldA, const float* B, unsigned ldB, float* C, unsigned ldC) {
  unsigned gx = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned gy = blockDim.y * blockIdx.y + threadIdx.y;
  if (gx < w && gy < h)
    C[gy*ldC + gx] = A[gy*ldA + gx] - B[gy*ldB + gx];
}

extern "C" void cuda_floatmatrix_sub(unsigned w, unsigned h, const float* A, unsigned ldA, const float* B, unsigned ldB, float* C, unsigned ldC) {

  dim3 grid((w + 15) / 15, (h + 15) / 15, 1);
  dim3 block(16,16,1);
  FloatMatrixSub<<<grid,block>>>(w, h, A, ldA, B, ldB, C, ldC);
}

/********** DUPLICATE **********/

__global__ void FloatMatrixDuplicate(unsigned w, unsigned h, const float* A, unsigned ldA, float* B, unsigned ldB) {

  unsigned gx = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned gy = blockDim.y * blockIdx.y + threadIdx.y;
  if (gx < w && gy < h)
    B[gy*ldB + gx] = A[gy*ldA + gx];
}

extern "C" void cuda_floatmatrix_duplicate(unsigned w, unsigned h, const float* A, unsigned ldA, float* B, unsigned ldB) {

  dim3 grid((w + 15) / 15, (h + 15) / 15, 1);
  dim3 block(16,16,1);
  FloatMatrixDuplicate<<<grid,block>>>(w, h, A, ldA, B, ldB);
}


/********** SUB MATRIX **********/

__global__ void FloatMatrixSubMatrix(unsigned x, unsigned y, unsigned w, unsigned h, const float* A, unsigned ldA, float* B, unsigned ldB) {

  unsigned gx = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned gy = blockDim.y * blockIdx.y + threadIdx.y;
  if (gx < w && gy < h)
    B[gy*ldB + gx] = A[(gy+y)*ldA + gx + x];
}

extern "C" void cuda_floatmatrix_submatrix(unsigned x, unsigned y, unsigned w, unsigned h, const float* A, unsigned ldA, float* B, unsigned ldB) {

  dim3 grid((w + 15) / 15, (h + 15) / 15, 1);
  dim3 block(16,16,1);
  FloatMatrixSubMatrix<<<grid,block>>>(x, y, w, h, A, ldA, B, ldB);
}


/********** SET **********/

__global__ void FloatMatrixSet(unsigned w, unsigned h, float value, float *A, unsigned ldA) {
  unsigned gx = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned gy = blockDim.y * blockIdx.y + threadIdx.y;
  if (gx < w && gy < h)
    A[gy*ldA + gx] = value;
}

extern "C" void cuda_floatmatrix_set(unsigned w, unsigned h, float value, float* A, unsigned ldA) {

  dim3 grid((w + 15) / 15, (h + 15) / 15, 1);
  dim3 block(16,16,1);
  FloatMatrixSet<<<grid,block>>>(w, h, value, A, ldA);
}

/********** TRANSPOSE **********/

__global__ void FloatMatrixTranspose(unsigned w, unsigned h, float *A, unsigned ldA, float *B, unsigned ldB) {
  unsigned gx = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned gy = blockDim.y * blockIdx.y + threadIdx.y;
  if (gx < w && gy < h)
    B[gx*ldB + gy] = A[gy*ldA+gx];
}

extern "C" void cuda_floatmatrix_transpose(unsigned w, unsigned h, float* A, unsigned ldA, float* B, unsigned ldB) {

  dim3 grid((w + 15) / 15, (h + 15) / 15, 1);
  dim3 block(16,16,1);
  FloatMatrixTranspose<<<grid,block>>>(w, h, A, ldA, B, ldB);
}

